/*
 * simple CUDA example to perform C=A*B matrix multiplication on GPU
 * 
 * For simplicity, value of C[i,j] is calculated by a thread from a 2D block. This may not be most efficient implementation!
 *
 * Each matrix is N*N
 *
 * mkbane (Dec2023)
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define DEFAULT_N 1024

// function prototypes
int fillRandom(float*, int);
int fillZeroes(float*, int);
void print2Darray(float*, int);

// CUDA kernel which runs on the GPU and is called from the host CPU
__global__ void myMM(float *C, float *A, float *B, int N) {
  /*
   * This kernel forms C=A*B where each matrix is N*N
  * NB various ways to do a 2D array in the C programming language.
  * This example flattens the array and then calcs relevant position 
  * in the array from the (row, col) indices
  *
  * The host CPU launches many instances of this kernel each running on a GPU thread
  * We are expecting 2D grid of blocks, with each block having 2D grid of threads
  * So, each thread has a unique thread index (threadIdx.x and threadIdx.y)
  * for each unique block index (blockIdx.x and blockIdx.y)
  * From thesse we can assign work to calc C[i,j]
  *
  */

  // rows are y-coords, cols are x-coords
  int myRow = (blockIdx.y * blockDim.y) + threadIdx.y;
  int myCol = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (myRow<N && myCol<N) {
    // unlike serial version, we do NOT loop over rows, cols
    // (since we have one thread for each (row,col) combination)
    int pos_ij = myRow+N*myCol;
    //DEBUG: printf("thread: myRow,myCol = %d,%d calculating C[%d]\n", myRow, myCol, pos_ij);
    int pos_ik, pos_kj;
    for (int k=0; k<N; k++) {
      pos_ik = myRow+N*k;
      pos_kj = k+N*myCol;
      //DEBUG: printf("to calc C[%d] += A[%d] * B[%d]\n adding %f * %f to %f\n",pos_ij,pos_ik,pos_kj,A[pos_ik],B[pos_kj],C[pos_ij]);
      C[pos_ij] += A[pos_ik]*B[pos_kj];
    }
    //DEBUG: printf("thread: C[%d]=%f\n", pos_ij, C[pos_ij]);
  }
  else {
    //DEBUG: printf("this thread is skipping (myRow=%d, myCol=%d; c.f. N=%d\n", myRow, myCol, N);
  }
  // at this point C[i,j] for all valid (i,j) has been calculated
} // CUDA kernel myMM


// host CPU cost that launches multiply CUDA kernels
int main(int argc, char *argv[]) {
  int N;
  if(argc>1) {
    N = atoi(argv[1]);
    printf("Each array is %d by %d\n", N,N);
    if(argc>2) printf("(ignoring other parameters)");
  }
  else {
    N = DEFAULT_N;
    printf("Each array is %d by %d\n", N,N);
  }

  // allocate and initialise
  float *A, *B, *C;
  int arrayMemSize=sizeof(float)*N*N;
  A = (float*) malloc(arrayMemSize);
  B = (float*) malloc(arrayMemSize);
  C = (float*) malloc(arrayMemSize);
  // check none of these failed
  if (A==NULL || B==NULL || C==NULL) {
    printf("failure to allocate arrays\n");
    abort();
  }
  printf("malloc complete\n");

  // fix seed of PRNG
  srand(101);
  int rc_A = fillRandom(A, N);
  int rc_B = fillRandom(B, N);
  int rc_C = fillZeroes(C, N);
  // check none of these failed
  if (rc_A != 0 || rc_B != 0 || rc_C !=0) {
    printf("failure to allocate arrays\n");
    abort();
  }

  printf("init complete\n");
#ifdef VERBOSE
  print2Darray(A, N);
  print2Darray(B, N);
#endif

  // declare memory used on GPU
  // we use dev_VARNAME to indicate variable VARNAME on the device (i.e. GPU)
  float *dev_A, *dev_B, *dev_C;
  hipMalloc(&dev_A, arrayMemSize);
  hipMalloc(&dev_B, arrayMemSize);
  hipMalloc(&dev_C, arrayMemSize);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) printf("error [post cudaMalloc]\n%s\n", hipGetErrorString(err));

  // copy values in each of A and B from host (CPU) too device (GPU)
  hipMemcpy(dev_A, A, arrayMemSize, hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, arrayMemSize, hipMemcpyHostToDevice);
  hipMemcpy(dev_C, C, arrayMemSize, hipMemcpyHostToDevice); // although would be best to zero on the GPU itself
  err = hipGetLastError();
  if (err != hipSuccess) printf("error [post memcpy to dev]\n%s\n", hipGetErrorString(err));

  // 2D grid. symmetrical. For each extent we have N data points.
  // Let's set tpb (threads-per-block) for each extent to be 256.
  // We therefore need, for each extent, N/tpb blocks but need to ensure #blocks * #threads >= N
  int tpb = 32;
  int nBlocks = ceil( (float) N / (float) tpb);

  dim3 blocks2D(nBlocks, nBlocks, 1);
  dim3 tpb2D(tpb, tpb, 1);
  printf("For each extent:\n using %d blocks each of %d threads. Total #threads=%d c.f. N=%d\n", nBlocks, tpb, nBlocks*tpb, N);
  printf("Total threads on device will be %d\n", (nBlocks*nBlocks)*(tpb*tpb));
  myMM<<<blocks2D, tpb2D>>>(dev_C, dev_A, dev_B, N);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error [post kernel] in CUDA kernel\n%s\n", hipGetErrorString(err));
  }
  else {
    hipDeviceSynchronize();
    // get result back onto host
    hipMemcpy(C, dev_C, arrayMemSize, hipMemcpyDeviceToHost); 
    #ifdef VERBOSE
    printf("matrix C (final answer)");
    print2Darray(C, N);
    #else
    // output given element of C
    //printf("C(%d,%d)=%f\n", N/2, N/2, C[N/2 + N * N/2]);
    printf("C(%d,%d)=%f\n", N/2, N/2, C[N/2,N/2]);
    #endif
  }

  // clean up and reset GPU
  free(A); free(B); free(C);
  hipFree(dev_A); hipFree(dev_B); hipFree(dev_C);
  hipDeviceReset();
  err = hipGetLastError();
  if (err != hipSuccess) printf("error [post cudaDeviceReset]\n%s\n", hipGetErrorString(err));


  printf("Finito!\n");
} // main

int fillRandom(float *array, int nSize) {
  // function to fill array with random numbers
  // note that rand returns integer so we divide by 1000 to get a float
  // and set zero as midpoint
  // where array is nSize * nSize
  // rc=0 upon success
  int rc = -1;
  for(int i=0; i<nSize*nSize; i++) {
    array[i] = (0.0001 * (float) rand()) - 0.00005*RAND_MAX;
  }
  rc = 0;
  return rc;
}

int fillZeroes(float *array, int nSize) {
  // function to fill array with zeroes
  // where array is nSize * nSize
  // rc=0 upon success
  int rc = -1;
  for(int i=0; i<nSize*nSize; i++) {
    array[i] = 0.00;
  }
  rc = 0;
  return rc;
}

void print2Darray(float *array, int nSize) {
  int row, col;
  // C stores data as col major
  for(int j=0; j<nSize; j++) {
    for(int i=0; i<nSize; i++) {
      printf("array[%d][%d]=%f\n",i,j,array[i+nSize*j]);
    }
  }
}
